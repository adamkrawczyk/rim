#include "hip/hip_runtime.h"
#include "AudioFir.cuh"
#include "hip/hip_runtime_api.h"
#include "stdio.h"

#define K 512

__global__ static void audiofir_kernel(float* yout, float* yin, float* coeff,
                                       int n, int len) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < len) {
    int j;
    float y_out = 0;
    for (j = 0; j <= n; j++) {
      if (i >= j) {
        y_out += yin[i - j] * coeff[j];
      }
    }
    yout[i] = y_out;
  }
}

void audiofir::audiofir(float* yout, float* yin, float* coeff, int n, int len,
                        ...) {
  checkCudaErrors(hipSetDevice(0));
  float *filter, *y_in, *y_out;
  checkCudaErrors(hipMalloc(&filter, (n + 1) * sizeof(float)));
  checkCudaErrors(hipMalloc(&y_in, (2 * len) * sizeof(float)));
  checkCudaErrors(hipMalloc(&y_out, (2 * len) * sizeof(float)));
  checkCudaErrors(hipMemcpy(filter, coeff, (n + 1) * sizeof(float),
                             hipMemcpyHostToDevice));
  checkCudaErrors(
      hipMemcpy(y_in, yin, (2 * len) * sizeof(float), hipMemcpyHostToDevice));

  hipEvent_t start, stop;  // pomiar czasu wykonania j?dra
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  checkCudaErrors(hipEventRecord(start, 0));
  audiofir_kernel<<<(len + K - 1) / K, K>>>(y_out, y_in, filter, n, len);
  checkCudaErrors(hipGetLastError());

  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  float elapsedTime;
  checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  checkCudaErrors(hipDeviceSynchronize());
  audiofir_kernel<<<(len + K - 1) / K, K>>>(y_out + len, y_in + len, filter, n,
                                            len);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMemcpy(yout, y_out, (2 * len) * sizeof(float),
                             hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree(filter));
  checkCudaErrors(hipFree(y_in));
  checkCudaErrors(hipFree(y_out));

  checkCudaErrors(hipDeviceReset());
  printf("GPU (kernel) time = %.3f ms (%6.3f GFLOP/s)\n", elapsedTime,
         1e-6 * 2 * ((double)n + 1) * 2 * ((double)len) / elapsedTime);
}
